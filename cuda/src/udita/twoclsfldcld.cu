#include "hip/hip_runtime.h"
#include "../explore.h"
#include "../mains.h"
#include "twoclsfldcld.h"

#define OUTPUT_BUFFER 1000000000

__device__ int8_t dci_buffer[OUTPUT_BUFFER];

// how many fields are in the DCI struct
#define STRUCT_FIELDS 5

void printDci(int8_t *dci_arr, int valid_dci, int struct_size) {
    printf("Programs:\n");
    for (int i = 0; i < valid_dci; i++) {
        for (int j = 0; j < struct_size; j++) {
            printf("%d ", dci_arr[(i * struct_size) + j]);
        }
        printf("\n");
    }
}

__device__ void dciStore(DCI *dci) {
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int32_t offset = idx * STRUCT_FIELDS;
    dci_buffer[offset] = dci->f_type; 
    dci_buffer[offset + 1] = dci->f_access_mod; 
    dci_buffer[offset + 2] = dci->is_static; 
    dci_buffer[offset + 3] = dci->f_access_type; 
    dci_buffer[offset + 4] = dci->subcls_rel; 
}

__device__ void dciGenerate(DCI *dci) {
    dci->f_type = (int8_t) _choice(0, MAX_FIELD_TYPE - 1);
    dci->f_access_mod = (int8_t) _choice(0, MAX_FIELD_ACCESS_MOD - 1);
    dci->is_static = (int8_t) _choice(0, IS_STATIC_CHOICE - 1);
    dci->f_access_type = (int8_t) _choice(0, MAX_FIELD_ACCESS_TYPE - 1);
    dci->subcls_rel = (int8_t) _choice(0, MAX_SUBCLS_REL - 1);
}

__global__ void twoClsFldCldUdita(int32_t bck_active, int32_t size) {
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= bck_active) {
        return;
    }

    DCI dci = {
        .f_type = 0,
        .f_access_mod = 0,
        .is_static = 0,
        .f_access_type = 0, 
        .subcls_rel = 0,
    };

    dciGenerate(&dci);
    dciStore(&dci);
    _countIf(TRUE);
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        printf("Incorrect arguments: size\n");
        exit(1);
    }
    int32_t size = atoi(argv[1]);

    // establish context
    hipFree(0);

    float time;
    hipEvent_t start, stop;

    CUDA_CHECK_RETURN(hipEventCreate(&start));
    CUDA_CHECK_RETURN(hipEventCreate(&stop));

    CUDA_CHECK_RETURN(hipEventRecord(start));
    int* args = (int*) malloc(sizeof(int));
    args[0] = size;

    explore((void (*)(...)) twoClsFldCldUdita, args, 1);
    // EXPLORE(twoClsFldCldUdita<<<starting_blocks, starting_threads>>>(active_threads, size));
    CUDA_CHECK_RETURN(hipEventRecord(stop));
    CUDA_CHECK_RETURN(hipEventSynchronize(stop));

    CUDA_CHECK_RETURN(hipEventElapsedTime(&time, start, stop));
    printf("Driver time: %.2lf\n", time);

    int valid_dci = *(_bck_stats->if_counter);
    int8_t *dci_arr = (int8_t*) calloc(valid_dci * STRUCT_FIELDS, sizeof(int8_t));
    CUDA_CHECK_RETURN(hipMemcpyFromSymbol(dci_arr, HIP_SYMBOL(dci_buffer), sizeof(int8_t) * valid_dci * STRUCT_FIELDS, 0, hipMemcpyDeviceToHost));
    
    printDci(dci_arr, valid_dci, STRUCT_FIELDS);

    free(dci_arr);

    // the following line if we use cuda-memcheck --leak-check
    hipDeviceReset();
}