#include "hip/hip_runtime.h"
#include <stdio.h>

#include "ptr.h"
#include "../explore.h"

ImpObjPool(Node);

#define OUTPUT_BUFFER 1000000000

__device__ int8_t adjacency_matrices[OUTPUT_BUFFER];

// Offset for adjacency_matrices. If update is set to TRUE,
// then offset will be updated at kernel launch.
__device__ int32_t offset = 0;

// The maximum thread id that successfully generated an inheritance
// graph during the last kernel invocation.
__device__ int32_t max_id = 0;

// Is set to true if a single thread was able to generate
// an inheritance graph.
__device__ int8_t update = FALSE;

void printPgs(int8_t *callgraphs, int valid_cgs, int adj_matrix_size) {
    printf("Programs:\n");
    for (int i = 0; i < valid_cgs; i++) {
        for (int j = 0; j < adj_matrix_size; j++) {
            printf("%d ", callgraphs[(i * adj_matrix_size) + j]);
        }
        printf("\n");
    }
}

void extractPgs(int8_t *source, int8_t *destination, int num_of_cgs, int matrix_size) {
    int destination_index = 0;

    // iterate over each adjacency matrix
    for (int i = 0; i < num_of_cgs; i++) {
        int source_index = i * (matrix_size + 1);
        if (source[source_index] == FALSE) {
            continue;
        }

        memcpy(&destination[destination_index], &source[source_index + 1], matrix_size * sizeof(int8_t));
        destination_index += matrix_size;
    }
}

__device__ int8_t nodeProperty(Node *n, LinkedList *path, Set *visited) {
    assert(path->size == 0);

    LinkedList work;
    llInit(&work);
    llAdd(&work, n);

    while (work.size != 0) {
        Node *current = llRemoveLast(&work);
        if (current == NULL) {
            llRemoveLast(path);
            continue;
        }

        // if not acyclic
        if (llContains(path, current)) {
            return FALSE;
        }

        llAdd(path, current);
        llAdd(&work, NULL);
        setAdd(visited, current);

        // check for diamond
        for (int32_t i = 0; i < current->num_of_children; i++) {
            Node *child = current->children[i];
            for (int32_t j = 0; j < i; j++) {
                if (child == current->children[j]) {
                    return FALSE;
                }
            }
        }

        // add all children
        for (int32_t i = 0; i < current->num_of_children; i++) {
            if (current->children[i] != NULL) {
                llAdd(&work, current->children[i]);
            }
        }
    }

    return TRUE;
}

__device__ int8_t nodePropertyBuggy(Node *n, Set *path, Set *visited) {
    if (setContains(path, n)) {
        return FALSE;
    }

    setAdd(path, n);
    setAdd(visited, n);
    for (int32_t i = 0; i < n->num_of_children; i++) {
        Node *child = n->children[i];
        // two children of a DAG cannot be the same object
        for (int32_t j = 0; j < i; j++) {
            if (child == n->children[j]) {
                return FALSE;
            }
        }
        // check property for every child of this node
        if (child != NULL && !(nodePropertyBuggy(child, path, visited))) {
            return FALSE;
        }
    }

    setRemove(path, n);
    return TRUE;
}

__device__ int8_t noDupChildren(Env *env) {
    NodePool *op = env->op;

    for (int32_t i = 0; i < op->size; i++) {
        Node *n = op->getObject(op, i);
        if (n != NULL) {
            for (int32_t i = 0; i < n->num_of_children; i++) {
                Node *child = n->children[i];
                for (int32_t j = 0; j < i; j++) {
                    if (child == n->children[j]) {
                        return FALSE;
                    }
                }
            }
        }
    }

    return TRUE;
}

__device__ void pgPrint(Node *root, char graph[]) {
    LinkedList work;
    llInit(&work);
    llAdd(&work, root);
    int index = 0;

    while (work.size != 0) {
        Node *current = llRemoveLast(&work);
        if (current == NULL) {
            graph[index++] = 'N';
            graph[index++] = '-';
            graph[index++] = '[';
            graph[index++] = ']';
            graph[index++] = '-';
            continue;
        } else {
            graph[index++] = current->id + '0';
            graph[index++] = '-';
            if (current->num_of_children == 0) {
                graph[index++] = '[';
                graph[index++] = ']';
                graph[index++] = '-';
            }
        }

        // add all children
        for (int32_t i = 0; i < current->num_of_children; i++) {
            llAdd(&work, current->children[i]);
        }
    }
    graph[index++] = 'e';
    graph[index] = '\0';
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;

    printf("Graph of id %d: %s\n", idx, graph);
}

// Adds generated inheritance graph to adjacency_matrices
__device__ void pgAdjMatrix(Env *env) {
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;

    NodePool *op = env->op;
    int adj_matrix_size = (op->size * op->size) + 1;
    int starting_index = (offset + idx) * adj_matrix_size;
    adjacency_matrices[starting_index] = TRUE;

    for (int32_t i = 0; i < op->size; i++) {
        Node *n = op->getObject(op, i);
        if (n != NULL) {
            for (int32_t j = 0; j < n->num_of_children; j++) {
                int8_t child_id = 0;
                if (n->children[j] != NULL) {
                    child_id = n->children[j]->id;
                }

                // + 1 because the first element indicates whether a valid structure exists here
                int index = starting_index + (i * op->size) + child_id + 1;
                if (index >= OUTPUT_BUFFER) {
                    printf("ERROR: index %d is out of bounds\n", index);
                    asm("trap;");
                }

                // adjacency_matrices[index] = n->edge_type[j];
                adjacency_matrices[index]++;
            }
        }
    }
}

__device__ void pgGenerate(Env *env) {
    NodePool *op = env->op;

    for (int32_t i = 0; i < op->size; i++) {
        Node *n = op->getObject(op, i);
        if (n != NULL) {
            n->id = i;
            int32_t num_of_children = _choice(0, env->num_of_nodes - 1);
            n->num_of_children = num_of_children;
            for (int32_t j = 0; j < num_of_children; j++) {
                n->children[j] = op->getAny(op);
                // n->edge_type[j] = _choice(1, 3);
            }
        }
    }
}

__global__ void pgUdita(int32_t bck_active, int32_t size) {
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= bck_active) {
        return;
    }

    if (idx == 0 && update == TRUE) {
        offset = offset + max_id + 1;
        max_id = 0;
        update = FALSE;
    }

    NodePool op;
    initNodePool(&op, size, EXCLUDE_NULL);

    Env env = {
        .num_of_nodes = size,
        .op = &op,
    };

    pgGenerate(&env);
    int8_t no_dups = noDupChildren(&env);
    _countIf(no_dups);
    if (no_dups == TRUE) {
        pgAdjMatrix(&env);
        atomicMax(&max_id, idx);
        update = TRUE;
    }
    // if (is_dag) {
    //     char graph[50];
    //     dagPrint(dag.root, graph);
    // }
}

__global__ void testNodeProperty(int32_t bck_active) {
    LinkedList ll_path;
    Set path;
    setInit(&path, &ll_path);

    LinkedList ll_visited;
    Set visited;
    setInit(&visited, &ll_visited);

    Node node;
    nodePropertyBuggy(&node, &path, &visited);
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        printf("Incorrect arguments: size\n");
        exit(1);
    }
    int32_t size = atoi(argv[1]);
    int32_t adj_matrix_size = size * size;

    // establish context
    hipFree(0);

    float time;
    hipEvent_t start, stop;
    int8_t *result = (int8_t*) calloc(OUTPUT_BUFFER, sizeof(int8_t));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(adjacency_matrices), result, sizeof(int8_t) * OUTPUT_BUFFER, 0, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipEventCreate(&start));
    CUDA_CHECK_RETURN(hipEventCreate(&stop));

    int* args = (int*) malloc(sizeof(int));
    args[0] = size;

    CUDA_CHECK_RETURN(hipEventRecord(start));
    explore((void (*)(...)) pgUdita, args, 1);
    CUDA_CHECK_RETURN(hipEventRecord(stop));
    CUDA_CHECK_RETURN(hipEventSynchronize(stop));

    CUDA_CHECK_RETURN(hipEventElapsedTime(&time, start, stop));
    printf("Driver time: %.2lf\n", time);

    int last_structure;
    int last_id;
    CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&last_structure, HIP_SYMBOL(offset), sizeof(int32_t), 0, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&last_id, HIP_SYMBOL(max_id), sizeof(int32_t), 0, hipMemcpyDeviceToHost));
    
    int result_size = last_structure + last_id + 1;
    CUDA_CHECK_RETURN(hipMemcpyFromSymbol(result, HIP_SYMBOL(adjacency_matrices), sizeof(int8_t) * (result_size) * (adj_matrix_size + 1), 0, hipMemcpyDeviceToHost));
    
    int valid_pgs = *(_bck_stats->if_counter);
    int8_t *pointer_graphs = (int8_t*) calloc(valid_pgs * adj_matrix_size, sizeof(int8_t));
    extractPgs(result, pointer_graphs, result_size, adj_matrix_size);

    printPgs(pointer_graphs, valid_pgs, adj_matrix_size);

    free(pointer_graphs);
    free(result);

    // the following line if we use cuda-memcheck --leak-check
    hipDeviceReset();
    free(args);

    return 0;
}
