#include "hip/hip_runtime.h"
#include "../explore.h"
#include "../mains.h"
#include "twoclsmtdcld.h"

#define OUTPUT_BUFFER 1000000000

__device__ int8_t dci_buffer[OUTPUT_BUFFER];

// how many fields are in the DCI struct
#define STRUCT_FIELDS 6

void printDci(int8_t *dci_arr, int valid_dci, int struct_size) {
    printf("Programs:\n");
    for (int i = 0; i < valid_dci; i++) {
        for (int j = 0; j < struct_size; j++) {
            printf("%d ", dci_arr[(i * struct_size) + j]);
        }
        printf("\n");
    }
}

__device__ void dciStore(DCI *dci) {
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int32_t offset = idx * STRUCT_FIELDS;
    dci_buffer[offset] = dci->f_modifier; 
    dci_buffer[offset + 1] = dci->m_access_mod; 
    dci_buffer[offset + 2] = dci->f_assignment_access_type; 
    dci_buffer[offset + 3] = dci->m_rel; 
    dci_buffer[offset + 4] = dci->m_access_type; 
    dci_buffer[offset + 5] = dci->subcls_rel; 
}

__device__ void dciGenerate(DCI *dci) {
    dci->f_modifier = (int8_t) _choice(0, MAX_FIELD_MODIFIER - 1);
    dci->m_access_mod = (int8_t) _choice(0, MAX_MTD_MODIFIER - 1);
    dci->f_assignment_access_type = (int8_t) _choice(0, MAX_FIELD_ACCESS_TYPE - 1);
    dci->m_rel = (int8_t) _choice(0, MAX_MTD_REL - 1);
    dci->m_access_type = (int8_t) _choice(0, MAX_MTD_ACCESS_TYPE - 1);
    dci->subcls_rel = (int8_t) _choice(0, MAX_SUBCLASS_REL - 1);
}

__global__ void twoClsMtdCldUdita(int32_t bck_active, int32_t size) {
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= bck_active) {
        return;
    }

    DCI dci = {
        .f_modifier = 0,
        .m_access_mod = 0,
        .f_assignment_access_type = 0,
        .m_rel = 0,
        .m_access_type = 0,
        .subcls_rel = 0,
    };

    dciGenerate(&dci);
    dciStore(&dci);
    _countIf(TRUE);
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        printf("Incorrect arguments: size\n");
        exit(1);
    }
    int32_t size = atoi(argv[1]);

    // establish context
    hipFree(0);

    float time;
    hipEvent_t start, stop;

    CUDA_CHECK_RETURN(hipEventCreate(&start));
    CUDA_CHECK_RETURN(hipEventCreate(&stop));

    CUDA_CHECK_RETURN(hipEventRecord(start));
    int* args = (int*) malloc(sizeof(int));
    args[0] = size;

    explore((void (*)(...)) twoClsMtdCldUdita, args, 1);
    // EXPLORE(twoClsMtdCldUdita<<<starting_blocks, starting_threads>>>(active_threads, size));
    CUDA_CHECK_RETURN(hipEventRecord(stop));
    CUDA_CHECK_RETURN(hipEventSynchronize(stop));

    CUDA_CHECK_RETURN(hipEventElapsedTime(&time, start, stop));
    printf("Driver time: %.2lf\n", time);

    int valid_dci = *(_bck_stats->if_counter);
    int8_t *dci_arr = (int8_t*) calloc(valid_dci * STRUCT_FIELDS, sizeof(int8_t));
    CUDA_CHECK_RETURN(hipMemcpyFromSymbol(dci_arr, HIP_SYMBOL(dci_buffer), sizeof(int8_t) * valid_dci * STRUCT_FIELDS, 0, hipMemcpyDeviceToHost));
    
    printDci(dci_arr, valid_dci, STRUCT_FIELDS);

    free(dci_arr);

    // the following line if we use cuda-memcheck --leak-check
    hipDeviceReset();
}