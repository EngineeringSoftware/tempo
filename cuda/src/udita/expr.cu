#include "hip/hip_runtime.h"
#include "../explore.h"
#include "../mains.h"
#include "expr.h"

#define OUTPUT_BUFFER 1000000000

// how many fields are in the FS struct
#define STRUCT_FIELDS 11

__global__ void exprUdita(int32_t bck_active, int32_t size) {
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= bck_active) {
        return;
    }

    AssignmentExprGenerator gen;
    AssignmentExpr expr = gen.generate(size);
    char output[500];
    expr.to_string(output, 0);

    printf("%s\n", output);
    _countIf(TRUE);
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        printf("Incorrect arguments: size\n");
        exit(1);
    }
    int32_t size = atoi(argv[1]);

    // establish context
    hipFree(0);

    float time;
    hipEvent_t start, stop;

    CUDA_CHECK_RETURN(hipEventCreate(&start));
    CUDA_CHECK_RETURN(hipEventCreate(&stop));

    int* args = (int*) malloc(sizeof(int));
    args[0] = size;

    CUDA_CHECK_RETURN(hipEventRecord(start));
    printf("Programs: \n");
    explore((void (*)(...)) exprUdita, args, 1);
    CUDA_CHECK_RETURN(hipEventRecord(stop));
    CUDA_CHECK_RETURN(hipEventSynchronize(stop));

    CUDA_CHECK_RETURN(hipEventElapsedTime(&time, start, stop));
    printf("Driver time: %.2lf\n", time);

    // int valid_fss = *(_bck_stats->if_counter);
    // int8_t *function_specifiers = (int8_t*) calloc(valid_fss * size, sizeof(int8_t));
    // CUDA_CHECK_RETURN(hipMemcpyFromSymbol(function_specifiers, HIP_SYMBOL(fs_buffer), sizeof(int8_t) * valid_fss * size, 0, hipMemcpyDeviceToHost));
    
    // printFss(function_specifiers, valid_fss, size);

    // free(function_specifiers);

    // the following line if we use cuda-memcheck --leak-check
    hipDeviceReset();
    free(args);
}